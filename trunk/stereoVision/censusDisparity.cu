#include "hip/hip_runtime.h"
/*
 * CensusDisparity.cu
 *
 * Computes the depth map based on the census algorithm
 * input is the left and right stereo image in BW
 *
 *  Created on: 26/09/2011
 *      Author: kimbjerge
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, bmp utilities
#include "defs.h"
#include "BmpUtil.h"
#include "timer.h"

#define CALLOC calloc
#define COUNT_TABLE_BITS 256

static unsigned int timerCUDA = 0;

static void calc_table (int N, long int start, long int end, unsigned char *table, unsigned char count) {

  /*printf("%d\t%d\t%d\t%d\n",N,start,end,count);*/
  if (N == 1) {
    *(table + start) = count;

  } else {
    calc_table (N/2, start, start + (end-start+1)/2 - 1, table, count);
    calc_table (N/2, start + (end-start+1)/2, end, table, count + 1);
  }
} /* calc_table */

static void print_table(unsigned char *count_table, int N) {
  int i;
  for (i=0; i<N; i++)
    printf("%d\t%d\n",i,*(count_table+i));
}

static void census_transform (unsigned char *image, int x_window_size, int y_window_size, int width, int height, int num_buffs, int size_buff, long int *census_tx) {
  int i, j, x_surround, y_surround, top, bottom, left, right, x, y, incr, index, k;
  unsigned char *image_row_ptr, *pix_ptr, *top_corner, centre_val;
  long int *census_ptr;

  printf ("CensusTransform [%d,%d], [%d,%d], %d, %d\n",x_window_size, y_window_size, width, height, num_buffs, size_buff);

  x_surround = (x_window_size - 1) / 2;
  y_surround = (y_window_size - 1) / 2;
  top = y_surround;
  left = x_surround;
  right = width - x_surround;
  bottom = height - y_surround;
  incr = width - x_window_size;

  image_row_ptr = image;

  for (y = top; y < bottom; y++) {
    census_ptr = census_tx + (y * width + left) * num_buffs;
    top_corner = image_row_ptr;

    for (x = left; x < right; x++) {
      pix_ptr =  top_corner;
      centre_val = *(top_corner + width * y_surround + x_surround);

      /* initialise census transform to 0 */
      for (i = 0; i < num_buffs; i++)
	    *(census_ptr + i) = 0;

      k = 0;
      for (i = 0; i < y_window_size; i++) {
		for (j = 0; j < x_window_size; j++) {
		  index = k / size_buff;
		  *(census_ptr + index) <<= 1;
		  if (*pix_ptr < centre_val)
			*(census_ptr + index) |= 1;
		  pix_ptr++;
		  k++;
		} /* for j */
		pix_ptr += incr;
      } /* for i */

      top_corner++;
      census_ptr += num_buffs;
    } /* for x */

    image_row_ptr += width;
  } /* for */
} /* census_transform */

void CENSUS_RIGHT_CUDA (unsigned char *left_image, unsigned char *right_image, signed char *disparity, double *min_array,
		           int width, int height, int x_census_win_size, int y_census_win_size, int x_window_size, int y_window_size, int min_disparity, int max_disparity) {
  unsigned int right_x;
  int right_lim, left_lim, y, i, top, bottom, left, right, x_surround, y_surround, diff, num_buffs, extra_bits, size_buff, div_buffs, u, v, incr, x_surr1, y_surr1;
  long int *census_left, *census_right, *ptr_censusl, *ptr_censusr, census_l, census_r, *buff_r, *buff_l, *lptr, *rptr, xor_res;
  int disp;

  unsigned char *count_table;

  count_table = (unsigned char*) CALLOC(256, sizeof(unsigned char));
  calc_table (COUNT_TABLE_BITS, 0, COUNT_TABLE_BITS-1, count_table, 0);
  /* print_table(count_table,COUNT_TABLE_BITS); */

  size_buff = sizeof(long int) * 8; // 32
  div_buffs = (x_census_win_size * y_census_win_size) / size_buff;
  extra_bits = (x_census_win_size * y_census_win_size) % size_buff;
  num_buffs = div_buffs + ((extra_bits > 0)?1:0);

  buff_l = (long int*) CALLOC(num_buffs, sizeof(long int));
  buff_r = (long int*) CALLOC(num_buffs, sizeof(long int));

  census_left = (long int*) CALLOC(width * height * num_buffs, sizeof(long int));
  census_transform (left_image, x_census_win_size, y_census_win_size, width, height, num_buffs, size_buff, census_left);
  census_right = (long int*) CALLOC(width * height * num_buffs, sizeof(long int));
  census_transform (right_image, x_census_win_size, y_census_win_size, width, height, num_buffs, size_buff, census_right);

  x_surround = (x_window_size - 1) / 2;
  y_surround = (y_window_size - 1) / 2;
  x_surr1 = x_surround + x_census_win_size/2;
  y_surr1 = y_surround + y_census_win_size/2;
  top = y_surr1;
  left = x_surr1;
  right = width - x_surr1;
  bottom = height - y_surr1;
  incr = (width - x_window_size) * num_buffs;

  /* Set minimum array to a really large number */
  for (i = 0; i < width * height; i++)
    min_array[i] = 1E10;

  for (disp = min_disparity; disp < max_disparity; disp++) {

	printf ("Disparity %d\n",disp);

    for (y = top; y < bottom; y++) {

      if (disp < 0) {
		ptr_censusl =  census_left + ((y - y_surround) * width + x_surr1 - x_surround) * num_buffs;
		ptr_censusr = census_right + ((y - y_surround) * width - disp + x_surr1 - x_surround) * num_buffs;

      } else {
		ptr_censusl =  census_left + ((y - y_surround) * width + disp + x_surr1 - x_surround) * num_buffs;
		ptr_censusr = census_right + ((y - y_surround) * width + x_surr1 - x_surround) * num_buffs;
      }

      right_lim = (disp < 0)? right : right - disp;
      left_lim = (disp < 0)? left - disp : left;
      /*printf("%d\n",y);*/

      for (right_x = left_lim; right_x < right_lim; right_x++) {

		lptr = ptr_censusl;
		rptr = ptr_censusr;

		diff = 0;
		for (u = 0; u < y_window_size; u++) {
		  for (v = 0; v < x_window_size * num_buffs; v++) {

			census_l = *lptr;
			census_r = *rptr;

			xor_res = census_l ^ census_r;
			for (i = 0; i < sizeof(long int); i++) {
			  diff += *(count_table + (xor_res & 0x00ff));
			}

			lptr ++;
			rptr ++;
		  } /* for v */

		  lptr += incr;
		  rptr += incr;
		} /* for u */

		if (diff < *(min_array + width * y + right_x)) {
		  *(disparity + width * y + right_x) = (unsigned char) disp; /* - min_disparity; */
		  *(min_array + width * y + right_x) = diff;
		} /* if */

		ptr_censusl += num_buffs;
		ptr_censusr += num_buffs;
      } /* for right_x */
    } /* for y*/
  } /* for disparity */

  free (count_table);
  free (buff_l); free(buff_r);
  free (census_left); free(census_right);

  printf("Census Right completed\n");
} /* CENSUS_RIGHT */

__global__ static void
average3DImages (byte* dst, int stride, hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
	  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
	  byte* imgPtr = (byte *)devPitchedPtr.ptr;
	  size_t pitch = devPitchedPtr.pitch;
	  size_t slicePitch = pitch * height;
	  byte pixel[DEPTH];

	  for (int z = 0; z < depth; ++z)
	  {
		  byte *slice = imgPtr + z * slicePitch; // Find sliced image
		  byte *row = slice + rowIdx * pitch; // Find row in image
		  pixel[z] = row[colIdx]; // Create array with pixel in both images
	  }

	  // Update average of stereo images
	  dst[rowIdx * stride + colIdx] = (pixel[0] + pixel[1])/2;
	  //dst[rowIdx * stride + colIdx] = pixel[1]; // 0 = left image, 1 = rigth image
}

__device__ static void swap (byte *x, byte *y)
{
	byte tmp;
	tmp = *x;
	*x = *y;
	*y = tmp;
}

__device__ static void bublesort (byte *a, int depth)
{
	int i, j;
	for (i = 0; i < (depth-1); i++)
		for (j = 0; j < (depth-(i+1)); j++)
			if (a[j] > a[j+1])
				swap(&a[j], &a[j+1]);
}


__global__ static void
median3DImages (byte* dst, int stride, hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
	  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
	  byte* imgPtr = (byte *)devPitchedPtr.ptr;
	  size_t pitch = devPitchedPtr.pitch;
	  size_t slicePitch = pitch * height;
	  byte median[DEPTH];

	  for (int z = 0; z < depth; ++z)
	  {
		  byte *slice = imgPtr + z * slicePitch; // Find sliced image
		  byte *row = slice + rowIdx * pitch; // Find row in image
		  median[z] = row[colIdx];
	  }

	  bublesort(median, depth);

	  dst[rowIdx * stride + colIdx] = median[(DEPTH+1)/2];
}

// Find depth map image based on 3D cube of images representing left and right images
float CensusDisparity(byte *ImgDst, byte *ImgSrc, ROI Size, int Stride, int depth,
		              int x_census_win_size, int y_census_win_size, int x_window_size, int y_window_size, int min_disparity, int max_disparity)
{
    byte *Dst;
    size_t DstStride;
    hipMemcpy3DParms memcpy3DParms = {0};

    DEBUG_MSG("[CensusDisparity]\n");

    // Create src pointer and extent
    memcpy3DParms.srcPtr = make_hipPitchedPtr(ImgSrc, Stride, Size.width, Size.height);
    memcpy3DParms.extent = make_hipExtent(Size.width * sizeof(byte), Size.height, depth);

    // Allocation of memory for 3D source images in byte format
    cutilSafeCall(hipMalloc3D(&memcpy3DParms.dstPtr, memcpy3DParms.extent));

    DEBUG_MSG("srcPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.srcPtr.pitch, memcpy3DParms.srcPtr.xsize, memcpy3DParms.srcPtr.ysize);
    DEBUG_MSG("dstPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.dstPtr.pitch, memcpy3DParms.dstPtr.xsize, memcpy3DParms.dstPtr.ysize);

    // Copy images to device memory
    memcpy3DParms.kind = hipMemcpyHostToDevice;
    cutilSafeCall(hipMemcpy3D(&memcpy3DParms));

    // Allocation of memory for 2D destination image in byte format
    cutilSafeCall(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(byte), Size.height));

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid( ceil((float)Size.width / BLOCK_SIZE), ceil((float)Size.height / BLOCK_SIZE) );

    DEBUG_MSG("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    DEBUG_MSG("Threads in Block [%d,%d]\n", threads.x, threads.y);

    if (timerCUDA == 0) CreateTimer(&timerCUDA);
    RestartTimer(timerCUDA);

    //median3DImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);
    average3DImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);

    StopTimer(timerCUDA);

    cutilSafeCall(hipDeviceSynchronize());

    cutilSafeCall(hipMemcpy2D(ImgDst, Size.width * sizeof(byte),
                                Dst, DstStride * sizeof(byte),
                                Size.width * sizeof(byte), Size.height,
                                hipMemcpyDeviceToHost) );

    cutilSafeCall(hipFree(memcpy3DParms.dstPtr.ptr));

    return GetTimer(timerCUDA);;
}



