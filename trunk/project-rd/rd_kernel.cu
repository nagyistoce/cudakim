#include "hip/hip_runtime.h"
#include "rd_kernel.h"

#include <stdio.h>
#include <timer.h>

// CUDA timer definition
unsigned int timerCUDA = 0;

// global scope
// declare texture reference for 1D float texture
texture<float, 1> texU;
texture<float, 1> texV;

/*
 * Utility function to initialize U and V
*/
__host__
void initializeConcentrations(unsigned int width, unsigned int height, float *U, float *V) {
		float *_U = new float[width*height];
		float *_V = new float[width*height];

		int k = 0;
		int i, j;

		for (i = 0; i < width * height; ++i) {
			_U[k] = 1.0f;
			_V[k++] = 0.0f;
		}

		for (i = (0.48f)*height; i < (0.52f)*height; ++i) {
			for (j = (0.48f)*width; j < (0.52f)*width; ++j) {
				_U[ (i * width + j) ] = 0.5f;
				_V[ (i * width + j) ] = 0.25f;
			}
		}

		// Now perturb the entire grid. Bound the values by [0,1]
		for (k = 0; k < width * height; ++k) {
			if ( _U[k] < 1.0f ) {
				float rRand = 0.02f*(float)rand() / RAND_MAX - 0.01f;
				_U[k] += rRand * _U[k];
			}
			if ( _V[k] < 1.0f ) {
				float rRand = 0.02f*(float)rand() / RAND_MAX - 0.01f;
				_V[k] += rRand * _V[k];
			}
		}

		// Upload initial state U and V to the GPU
		hipMemcpy( U, _U, width*height*sizeof(float), hipMemcpyHostToDevice );
		hipMemcpy( V, _V, width*height*sizeof(float), hipMemcpyHostToDevice );

		delete[] _U;
		delete[] _V;
}

/*
 * Kernel for the reaction-diffusion model
 * This kernel is responsible for updating 'U' and 'V'
 */
__global__
void rd_kernel(unsigned int width, unsigned int height,
               float dt, float dx, float Du, float Dv,
               float F, float k, float *U, float *V) {

	// Coordinate of the current pixel (for this thread)
	const uint2 co = make_uint2( blockIdx.x*blockDim.x + threadIdx.x,
                                 blockIdx.y*blockDim.y + threadIdx.y );
	
	// Linear index of the curernt pixel
	const unsigned int idx = co.y*width + co.x;

	//
	// REACTION-DIFFUSION KERNEL - Kim Bjerge's version
	//
	// done - Notes: - optimization - kernel without "if"
	// done - Texture version for Mac
	// done - Meassurments - time
	// done - Ressourcer forbrug?

	// Tile assymetrisk ?
	// Kernel with shared memory how ?

        // Use registeres to save current values of U and V
        float Ui = U[idx];
        float Vi = V[idx];

        // Skip computing first and last line in image
        if (idx >= width && idx < width*(height-1))
        {
		// Computes the Laplacian operator for U and V - used values in x and y dimensions
		//float laplacianU = Ui;
		//float laplacianV = Vi;
		float laplacianU = (U[idx+1] + U[idx-1] + U[idx+width] + U[idx-width] - 4 * Ui)/(dx*dx);
		float laplacianV = (V[idx+1] + V[idx-1] + V[idx+width] + V[idx-width] - 4 * Vi)/(dx*dx);


		// Computes the diffusion and reaction of the two chemicals reactants mixed together
		float Uf = Du * laplacianU - Ui*powf(Vi,2) + F*(1 - Ui);
		//float Uf = Du * laplacianU; // Difusion only
		float Vf = Dv * laplacianV + Ui*powf(Vi,2) - (F + k)*Vi;

		U[idx] = Ui + dt*Uf;
		V[idx] = Vi + dt*Vf;
        }
        
}

/*
 * Optimized kernel for the reaction-diffusion model
 * Using texture memory for U and V
 * This kernel is responsible for updating 'U' and 'V'
 */
__global__
void rd_kernel_tex(unsigned int width, unsigned int height,
               float dt, float dx, float Du, float Dv,
               float F, float k, float *U, float *V) {

	// Coordinate of the current pixel (for this thread)
	const uint2 co = make_uint2( blockIdx.x*blockDim.x + threadIdx.x,
                                 blockIdx.y*blockDim.y + threadIdx.y );

	// Linear index of the curernt pixel
	const unsigned int idx = co.y*width + co.x;

	//
	// REACTION-DIFFUSION KERNEL - Kim Bjerge's version
	//

	// Use registeres to save current values of U and V

    float Ui = tex1Dfetch(texU, idx);
	float Vi = tex1Dfetch(texV, idx);

	// Skip computing first and last line in image
	if (idx >= width && idx < width*(height-1))
	{
		// Computes the Laplacian operator for U and V - used values in x and y dimensions
		float laplacianU = (tex1Dfetch(texU, idx+1) + tex1Dfetch(texU,idx-1) + tex1Dfetch(texU, idx+width) + tex1Dfetch(texU, idx-width) - 4 * Ui)/(dx*dx);
		float laplacianV = (tex1Dfetch(texV, idx+1) + tex1Dfetch(texV, idx-1) + tex1Dfetch(texV, idx+width) + tex1Dfetch(texV, idx-width) - 4 * Vi)/(dx*dx);


		// Computes the diffusion and reaction of the two chemicals reactants mixed together
		float Uf = Du * laplacianU - Ui*powf(Vi,2) + F*(1 - Ui);
		float Vf = Dv * laplacianV + Ui*powf(Vi,2) - (F + k)*Vi;

		U[idx] = Ui + dt*Uf;
		V[idx] = Vi + dt*Vf;
	}

}

/*
 * Kernel for the reaction-diffusion model
 * This kernel is responsible for updating 'U' and 'V'
 */
__global__
void rd_kernel_opt1(unsigned int width, unsigned int height,
               float dt, float dx, float Du, float Dv,
               float F, float k, float *U, float *V) {

	// Coordinate of the current pixel (for this thread)
	const uint2 co = make_uint2( blockIdx.x*blockDim.x + threadIdx.x,
                                 blockIdx.y*blockDim.y + threadIdx.y );

	// Linear index of the curernt pixel
	const unsigned int idx = co.y*width + co.x;

	//
	// REACTION-DIFFUSION KERNEL - Optimized version 1
	//

	// Use registeres to save current values of U and V
	float Ui = U[idx];
	float Vi = V[idx];

	// Computes the Laplacian operator for U and V - used values in x and y dimensions
	float laplacianU = (U[idx+1] + U[idx-1] + U[idx+width] + U[idx-width] - 4 * Ui)/(dx*dx);
	float laplacianV = (V[idx+1] + V[idx-1] + V[idx+width] + V[idx-width] - 4 * Vi)/(dx*dx);


	// Computes the diffusion and reaction of the two chemicals reactants mixed together
	float Uf = Du * laplacianU - Ui*powf(Vi,2) + F*(1 - Ui);
	float Vf = Dv * laplacianV + Ui*powf(Vi,2) - (F + k)*Vi;

	// Needed since U and V values used by all threads in block
	__syncthreads();

	U[idx] = Ui + dt*Uf;
	V[idx] = Vi + dt*Vf;
}

/*
 * Kernel for the reaction-diffusion model
 * This kernel is responsible for updating 'U' and 'V'
 */
__global__
void rd_kernel_opt2(unsigned int width, unsigned int height,
               float dt, float dx, float Du, float Dv,
               float F, float k, float *U, float *V) {

	// Coordinate of the current pixel (for this thread)
	const uint2 co = make_uint2( blockIdx.x*blockDim.x + threadIdx.x,
                                 blockIdx.y*blockDim.y + threadIdx.y );

	// Linear index of the curernt pixel
	const unsigned int idx = co.y*width + co.x;

	// REACTION-DIFFUSION KERNEL - Optimized version 2
	// Use registeres to save current values of U and V

	U[idx] = U[idx] + dt*(Du * ((U[idx+1] + U[idx-1] + U[idx+width] + U[idx-width] - 4 * U[idx])/(dx*dx)) - U[idx]*V[idx]*V[idx] + F*(1 - U[idx]));
	V[idx] = V[idx] + dt*(Dv * ((V[idx+1] + V[idx-1] + V[idx+width] + V[idx-width] - 4 * V[idx])/(dx*dx)) + U[idx]*V[idx]*V[idx] - (F + k)*V[idx]);
}

/*
 * Wrapper for the reaction-diffusion kernel. 
 * Called every frame by 'display'
 * 'result_devPtr' is a floating buffer used for visualization.
 * Make sure whatever needs visualization goes there.
 */
extern "C" __host__
void rd(unsigned int width, unsigned int height, float *result_devPtr) {
	// Create buffers for 'U' and 'V' at first pass
	static float *U, *V;
	static bool first_pass = true;

	if (first_pass){
		// Allocate device memory for U and V
		hipMalloc((void**)&U, width*height*sizeof(float));
		hipMalloc((void**)&V, width*height*sizeof(float));
 
		// Check for Cuda errors
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
			printf("\nCuda error detected: %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
			exit(1);
		}

		// Initialize U and V on the CPU and upload to the GPU
		initializeConcentrations( width, height, U, V );

		CreateTimer(&timerCUDA);

		// Make sure we never get in here again...
		first_pass = false;
	}

	// Kernel block dimensions
	const dim3 blockDim(16,16);

	// Verify input image dimensions
	if (width%blockDim.x || height%blockDim.y) {
		printf("\nImage width and height must be a multiple of the block dimensions\n");
		exit(1);
	}

	// Experiment with different settings of these constants
        /* Original values
	const float dt = 1.0f;
	const float dx = 2.0f;
	const float Du = 0.0004f*((width*height)/100.0f);
	const float Dv = 0.0002f*((width*height)/100.0f);
	const float F = 0.012f; 
	const float k = 0.052f;
        */

	const float dt = 0.2f;
	const float dx = 2.0f;
	const float Du = 0.0004f*((width*height)/100.0f);
	const float Dv = 0.0002f*((width*height)/100.0f); // Impact on how fast V diffusses (0.0001 or 0.0002)
	const float F = 0.012f; 
	const float k = 0.052f;


	// Invoke kernel (update U and V)
#if 1 // Optimized skipping top and bottom edges
	RestartTimer(timerCUDA);
	//rd_kernel<<< dim3(width/blockDim.x, height/blockDim.y), blockDim >>>( width, height, dt, dx, Du, Dv, F, k, U, V );
	//rd_kernel_opt1<<< dim3(width/blockDim.x, (height-2)/blockDim.y), blockDim >>>( width, height-2, dt, dx, Du, Dv, F, k, &U[width], &V[width] );
	rd_kernel_opt2<<< dim3(width/blockDim.x, (height-2)/blockDim.y), blockDim >>>( width, height-2, dt, dx, Du, Dv, F, k, &U[width], &V[width] );
	StopTimer(timerCUDA);
	float average = GetAverage(timerCUDA);
	if (average > 0)
	   printf("Opt2 %f ms\n", average);
#endif

#if 0 // Optimized with texture memory
    // Create texture for U matrix
    const hipChannelFormatDesc descU = hipCreateChannelDesc<float>();
    size_t numU_bytes = width*height*sizeof(float);
    hipBindTexture(NULL, &texU, (const void*)U, &descU, numU_bytes);

    // Create texture for V matrix
    const hipChannelFormatDesc descV = hipCreateChannelDesc<float>();
    size_t numV_bytes = width*height*sizeof(float);
    hipBindTexture(NULL, &texV, (const void*)V, &descV, numV_bytes);

    RestartTimer(timerCUDA);
	rd_kernel_tex<<< dim3(width/blockDim.x, height/blockDim.y), blockDim >>>( width, height, dt, dx, Du, Dv, F, k, U, V);
	StopTimer(timerCUDA);
	float average = GetAverage(timerCUDA);
	if (average > 0)
		printf("Tex %f ms\n", average);

    hipUnbindTexture(texU);
    hipUnbindTexture(texV);
#endif

	// Check for errors
	hipError_t err = hipGetLastError();
	if( err != hipSuccess ){
		printf("\nCuda error detected in 'rd_kernel': %s. Quitting.\n", hipGetErrorString(err) ); fflush(stdout);
		exit(1);
	}

	// For visualization we use a 'float1' image. You can use either 'U' or 'V'.
	hipMemcpy( result_devPtr, V, width*height*sizeof(float), hipMemcpyDeviceToDevice );
}
