#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, bmp utilities
#include "BmpUtil.h"

// includes, kernels
#include "image_kernel.cu"

/**
*  The dimension of pixels block 16x16
*/
#define BLOCK_SIZE			16

int g_TotalFailures = 0;


byte *NextImage(byte *pImage, int imgStride, ROI size)
{
	return (pImage + (imgStride*size.height));
}

// Loads image from file
// Allocates memory for source and destination of image 
// based size of image, image type must be bmp
int 
loadImages(char* fileName, const char* path, byte** imgSrc, byte** imgDst, ROI* imgSize, int *imgStride, int depth)
{
    //preload image (acquire dimensions)
    int ImgWidth, ImgHeight;
    byte *imgCur;
    //char *pImageFpath = cutFindFilePath(fileName, path);
    char ImageName[50];

    sprintf(ImageName, fileName, 1);

    int res = PreLoadBmp(ImageName, &ImgWidth, &ImgHeight);
    if (res)
    {
        printf("\nError %d: Image file %s not found or invalid!\n", res, ImageName);
        printf("Press ENTER to exit...\n");
        getchar();

        return 1;
    }

    //check image dimensions are multiples of BLOCK_SIZE
    if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0)
    {
        printf("\nError: Input image dimensions must be multiples of 8!\n");
        printf("Press ENTER to exit...\n");
        getchar();

        return 1;
    }

    //allocate image buffers
    *imgDst = MallocPlaneByte(ImgWidth, ImgHeight, imgStride);
    *imgSrc = MallocCubeByte(ImgWidth, ImgHeight, depth, imgStride);

    imgSize->width = ImgWidth;
    imgSize->height = ImgHeight;
    imgCur = *imgSrc;

    //load sample images
    for (int i = 1; i <= depth; i++)
    {
        printf("Loading image %s [%d,%d] \n", ImageName, ImgWidth, ImgHeight);
    	sprintf(ImageName, fileName, i);
    	res = PreLoadBmp(ImageName, &ImgWidth, &ImgHeight);
        if (res)
        {
            printf("\nError %d: Image file %s not found or invalid!\n", res, ImageName);
            printf("Press ENTER to exit...\n");
            getchar();
            return 1;
        }

    	LoadBmpAsGray(ImageName, *imgStride, *imgSize, imgCur);
    	imgCur = NextImage(imgCur, *imgStride, *imgSize);
    }

    
    printf("Images size [%d * %d * %d], %d \n", ImgWidth, ImgHeight, depth, *imgStride);
    
    return 0;
}

float ImageBackground(byte *ImgSrc, byte *ImgDst, ROI Size, int Stride, int depth)
{
    byte *Dst;
    size_t DstStride;
    hipMemcpy3DParms memcpy3DParms = {0};

    // Create src pointer and extent
    memcpy3DParms.srcPtr = make_hipPitchedPtr(ImgSrc, Stride, Size.width, Size.height);
    memcpy3DParms.extent = make_hipExtent(Size.width * sizeof(byte), Size.height, depth);

    // Allocation of memory for 3D source images in byte format
    cutilSafeCall(hipMalloc3D(&memcpy3DParms.dstPtr, memcpy3DParms.extent));

    printf("srcPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.srcPtr.pitch, memcpy3DParms.srcPtr.xsize, memcpy3DParms.srcPtr.ysize);
    printf("dstPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.dstPtr.pitch, memcpy3DParms.dstPtr.xsize, memcpy3DParms.dstPtr.ysize);

    // Copy images to device memory
    memcpy3DParms.kind = hipMemcpyHostToDevice;
    cutilSafeCall(hipMemcpy3D(&memcpy3DParms));

    // Allocation of memory for 2D destination image in byte format
    cutilSafeCall(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(byte), Size.height));

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid( ceil((float)Size.width / BLOCK_SIZE), ceil((float)Size.height / BLOCK_SIZE) );

    printf("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    printf("Threads in Block [%d,%d]\n", threads.x, threads.y);


    medianImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);
    //test3DImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);
    cutilSafeCall(hipDeviceSynchronize());

    printf("Copy result to host\n");
    cutilSafeCall(hipMemcpy2D(ImgDst, Size.width * sizeof(byte),
                                Dst, DstStride * sizeof(byte),
                                Size.width * sizeof(byte), Size.height,
                                hipMemcpyDeviceToHost) );

    cutilSafeCall(hipFree(memcpy3DParms.dstPtr.ptr));

    return 0;
}

float MorphEdge(byte *ImgSrc, byte *ImgDst, ROI Size, int Stride)
{    
    float *Dst, *DstBW, *Src, *Diff;
    size_t DstStride, SrcStride, DiffStride;
    
    //convert source image to float representation
    int ImgSrcFStride;
    float *ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);
    CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);
    
    // Allocation of memory for 2D source image in single precision format
    cutilSafeCall(hipMallocPitch((void **)(&Src), &SrcStride, Size.width * sizeof(float), Size.height));
    SrcStride /= sizeof(float);
    printf("SrcStride %d\n", SrcStride);

    //copy source image from host memory to device
    cutilSafeCall(hipMemcpy2D(Src, SrcStride * sizeof(float),
                               ImgSrcF, ImgSrcFStride * sizeof(float), 
                               Size.width * sizeof(float), Size.height,
                               hipMemcpyHostToDevice) );

    // Allocation of device memory for 2D destination image in single precision format
    cutilSafeCall(hipMallocPitch((void **)(&DstBW), &DstStride, Size.width * sizeof(float), Size.height));
    cutilSafeCall(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(float), Size.height));
    DstStride /= sizeof(float);
    
    cutilSafeCall(hipMallocPitch((void **)(&Diff), &DiffStride, Size.width * sizeof(float), Size.height));
    DiffStride /= sizeof(float);

    //setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    printf("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    printf("Threads in Block [%d,%d]\n", threads.x, threads.y);

    //create and start CUDA timer
    unsigned int timerCUDA = 0;
    cutilCheckError(cutCreateTimer(&timerCUDA));
    cutilCheckError(cutResetTimer(timerCUDA));
    cutilCheckError(cutStartTimer(timerCUDA));
    
    //copy image from device memory to device memory
    /*
    cutilSafeCall(hipMemcpy2D(Dst, DstStride * sizeof(float),  
                                Src, SrcStride * sizeof(float), 
                                Size.width * sizeof(float), Size.height,
                                hipMemcpyDeviceToDevice) );
    
    copyImage<<< grid, threads >>>(Dst, Src, Size.width);
    */

    // Generate BW image
    tresholdImage<<< grid, threads >>>(DstBW, Src, Size.width, 110);
    cutilSafeCall(hipDeviceSynchronize());

    // Dilate image with structuring element
    dilateImage<<< grid, threads >>>(Dst, DstBW, Size.width);
    // Erode image with structuring element
    //erodeImage<<< grid, threads >>>(Dst, DstBW, Size.width);
    cutilSafeCall(hipDeviceSynchronize());
    
    // Diff BW and eroded image
    diffImage<<< grid, threads >>>(Diff, DstBW, Dst, Size.width);
    cutilSafeCall(hipDeviceSynchronize());

    cutilCheckError(cutStopTimer(timerCUDA));

    cutilCheckMsg("Kernel execution failed");

    // finalize CUDA timer
    float TimerCUDASpan = cutGetAverageTimerValue(timerCUDA);
    cutilCheckError(cutDeleteTimer(timerCUDA));

    //copy eroded image from device memory to host memory in Src
    cutilSafeCall(hipMemcpy2D(ImgSrcF, ImgSrcFStride * sizeof(float), 
                                Diff, DiffStride * sizeof(float), 
                                Size.width * sizeof(float), Size.height,
                                hipMemcpyDeviceToHost) );
                                   
    CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

    //clean up memory
    cutilSafeCall(hipFree(Src));
    cutilSafeCall(hipFree(Dst));
    cutilSafeCall(hipFree(DstBW));
    cutilSafeCall(hipFree(Diff));
    FreePlane(ImgSrcF);

    //return time taken by the operation
    return TimerCUDASpan;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	byte *ImgSrc, *ImgDst;
	ROI ImgSize;
	int ImgStride;
    printf("[imageMorph]\n");
    int devID;
    int depth = DEPTH;

    //char ImageFname[] = "rice.bmp";
    //char ImageFname[] = "ricebw.bmp";
    char ImageFname[] = "data/E45nord%d.bmp";
    char EdgeImageFname[] = "nordEdge.bmp";

    hipDeviceProp_t deviceProps;

	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
	    devID = cutilDeviceInit(argc, argv);
            if (devID < 0) {
               printf("exiting...\n");
               cutilExit(argc, argv);
               exit(0);
            }
	}
	else {
	    devID = cutGetMaxGflopsDeviceId();
	    hipSetDevice( devID );
	}
		
    // get number of SMs on this GPU
    cutilSafeCall(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);
    
    // Load image and allocate memory
    if (loadImages(ImageFname, argv[0], &ImgSrc, &ImgDst, &ImgSize, &ImgStride, depth))
    {
        //finalize
        cutilExit(argc, argv);
        return 1;
    }
    
    // Test image - Rice black/white image
    printf("Image 0[0,%d], 10[0,%d], 11[255,%d], 22[255,%d], 23[0,%d], 256[0,%d]\n", 
            ImgSrc[256], ImgSrc[256*9], ImgSrc[256*10], ImgSrc[256*21], ImgSrc[256*22], ImgSrc[256*255]);

    //printf("Erode image\n");
    //float TimeCUDA1 = MorphEdge(ImgSrc, ImgDst, ImgSize, ImgStride);
    //printf("Processing time (ErodeCUDA 1)    : %f ms \n", TimeCUDA1);

    printf("Average image %d\n", ImgStride);
    float TimeCUDA1 =  ImageBackground(ImgSrc, ImgDst, ImgSize, ImgStride, depth);
    printf("Processing time (Background 1)    : %f ms \n", TimeCUDA1);
    
    //dump result of Gold 1 processing
    printf("Success\nDumping result to %s...\n", EdgeImageFname);
    DumpBmpAsGray(EdgeImageFname, ImgDst, ImgStride, ImgSize);

    //release byte planes
    FreePlane(ImgSrc);
    FreePlane(ImgDst);

    cutilExit(argc, argv);
}
