/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 /*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, bmp utilities
#include "defs.h"
#include "BmpUtil.h"
#include "timer.h"
#include "imageLoader.h"
#include "deviceUtil.h"

// includes, kernels and functions
#include "imageBackground.h"
#include "locateObjects.h"
#include "labelObjects.h"
#include "imageThrust.h"

static unsigned int timerTotalCUDA = 0;

/* Remaining work -
* OK - Color result images
* OK - Update input images using matlab - remove header time
* - Optimize labelObjects - reduction kernel
* - Run computeprof
* - Gausian bluring of diff images
* - Document results
*/

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	byte *ImgSrc, *ImgDst, *ImgDiff, *ImgBW, *ImgBack, *ImgCur;
	ROI ImgSize; // ImgBackSize;
	int ImgSrcStride, ImgDstStride, ImgBackStride, ImgBWStride;
    int devID;
    int depth = DEPTH;
    float TimeCUDA;
    float TimeTotal = 0;
    float TimeLableObjects = 0;
    char ImageName[50];
    int ObjectsFound, Objects;

    //char ImageFname[] = "rice.bmp";
    //char ImageBackFname[] = "nordBack.bmp";
    char ImageFname[] = "data/E45nord%d.bmp";
    char BackImageFname[] = "nordBackground.bmp";
    char TestImageFname[] = "nordResult%d.bmp";

    printf("ImageCount version 1.0\n")
    printf("Program counting objects in series of images\n");
    printf("--------------------------------------------------\n");

	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
	    devID = cutilDeviceInit(argc, argv);
		if (devID < 0) {
		   printf("exiting...\n");
		   cutilExit(argc, argv);
		   exit(0);
		}
	}
	else {
	    devID = cutGetMaxGflopsDeviceId();
	    hipSetDevice( devID );
	}
		
    // get number of SMs on this GPU
    PrintDeviceProperties();

    // Load images 1-9 and allocate memory
    if (loadImages(ImageFname, argv[0], &ImgSrc, &ImgSize, &ImgSrcStride, depth))
    {
        //finalize
        cutilExit(argc, argv);
        return 1;
    }
    
    // Initialize timer
    CreateTimer(&timerTotalCUDA);
    StartTimer(timerTotalCUDA);

    ImgBack = MallocPlaneByte(ImgSize.width, ImgSize.height, &ImgBackStride);

    //printf("Image src stride %d\n", ImgSrcStride);
    TimeCUDA = ImageBackground(ImgBack, ImgSrc, ImgSize, ImgSrcStride, depth);
    printf("Processing time (ImageBackground)    : %f ms \n", TimeCUDA);
    TimeTotal += TimeCUDA;

    // Save temporary background image in file
    //Dump result of finding background image
    printf("Dumping background image to %s...\n", BackImageFname);
    DumpBmpAsGray(BackImageFname, ImgBack, ImgBackStride, ImgSize);
    //------------------------------------------------------------------------------------------

    /*
    printf("--------------------------------------------------\n");
    // Testing of diff background with images
    // Load image and allocate memory
    if (loadImage(ImageBackFname, argv[0], &ImgBack, &ImgBackSize, &ImgBackStride))
    {
        //finalize
        cutilExit(argc, argv);
        return 1;
    }
    */

    // Allocate images
    ImgDst = MallocPlaneByte(ImgSize.width, ImgSize.height, &ImgDstStride);
    ImgDiff = MallocPlaneByte(ImgSize.width, ImgSize.height, &ImgBWStride);
    ImgBW = MallocPlaneByte(ImgSize.width, ImgSize.height, &ImgBWStride);

    printf("--------------------------------------------------\n");
    printf("Locating and label of objects based on background \n");

    ImgCur = ImgSrc;
    ObjectsFound = 0;
    for (int i = 1; i <= depth; i++)
    {
		TimeCUDA = DiffImages(ImgDiff, ImgBack, ImgCur, ImgSize, ImgSrcStride, ImgBackStride);
		//TimeCUDA = ThrustImageDiff(ImgDst, ImgBack, ImgCur, ImgSize, ImgSrcStride, ImgBackStride);
		printf("Processing time (DiffImages)      : %f ms \n", TimeCUDA);
	    TimeTotal += TimeCUDA;

	    ImgCur = NextImage(ImgCur, ImgSrcStride, ImgSize);

		TimeCUDA = MorphObjects(ImgBW, ImgDiff, ImgSize, ImgBWStride);
	    printf("Processing time (MorphObjects)    : %f ms \n", TimeCUDA);
	    TimeTotal += TimeCUDA;

	    TimeCUDA = LabelObjects(ImgDst, ImgBW, ImgSize, ImgDstStride, &Objects);
	    printf("Processing time (LabelObjects)    : %f ms \n", TimeCUDA);
	    TimeTotal += TimeCUDA;
	    TimeLableObjects += TimeCUDA;

	    ObjectsFound += Objects;

		sprintf(ImageName, TestImageFname, i);
		//printf("Dumping BW image to %s...\n", ImageName);
		//DumpBmpAsGray(ImageName, ImgBW, ImgBWStride, ImgSize);
		printf("Dumping Diff image to %s...\n", ImageName);
		DumpBmpAsGray(ImageName, ImgDiff, ImgBWStride, ImgSize);

		sprintf(ImageName, TestImageFname, i+10);
		printf("Dumping Label image to %s...\n", ImageName);
		DumpBmpColorMap(ImageName, ImgDst, ImgDstStride, ImgSize, redColorMap, RED_COLOR_MAP_SIZE);
		//DumpBmpAsGray(ImageName, ImgDst, ImgDstStride, ImgSize);
    }

    StopTimer(timerTotalCUDA);
    float time = GetTimer(timerTotalCUDA);
    printf("Image processing time (Total)     : %f ms \n", TimeTotal);
    printf("Image label objects time (Total)  : %f ms \n", TimeLableObjects);
    printf("Processing time (Total)           : %f ms \n", time);

    printf("Total number of objects found     : %d \n", ObjectsFound);

    //release byte planes
    FreePlane(ImgSrc);
 	FreePlane(ImgBack);
    FreePlane(ImgDst);
    FreePlane(ImgDiff);
    FreePlane(ImgBW);

    cutilExit(argc, argv);
}
