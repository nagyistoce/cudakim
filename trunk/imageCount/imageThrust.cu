#include "hip/hip_runtime.h"
/*
 * imageThrust.h
 *
 *  Created on: 26/09/2011
 *      Author: kimbjerge
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, thrust
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/adjacent_difference.h>
#include <thrust/random.h>

#include <iostream>
#include <iterator>

// includes, timer utilities
#include "defs.h"
#include "BmpUtil.h"
//#include "timer.h"

//static unsigned int timerCUDA = 0;

struct func_diff_byte
{
	__host__ __device__
	byte operator()(byte a, byte b)
	{
		return abs(a - b);
	}
};

// Computes difference between images
// COULD BE OPTIMIZED!
float ThrustImageDiff(byte *ImgDst, byte *ImgBack, byte *ImgSrc, ROI Size, int ISStride, int IBStride)
{
    hipEvent_t start;
    hipEvent_t end;
    float elapsed_time;
    int idx, ImgSize = ISStride*Size.height*sizeof(byte);

    DEBUG_MSG("[ThrustImageDiff]\n");

    hipEventCreate(&start);
    hipEventCreate(&end);
    //if (timerCUDA == 0) CreateTimer(&timerCUDA);
    hipEventRecord(start,0);

    thrust::host_vector<byte> hostImgBack(ImgSize);
    thrust::host_vector<byte> hostImgSrc(ImgSize);

    // How to copy ImgBack and ImgSrc to host_vectors efficient?
    for (idx = 0; idx < ImgSize; idx++)
    {
    	hostImgSrc[idx] = ImgSrc[idx];
    	hostImgBack[idx] = ImgBack[idx];
    }

    // Copy host vectors to devices
    thrust::device_vector<byte> devImgBack = hostImgBack;
    thrust::device_vector<byte> devImgSrc = hostImgSrc;
    thrust::device_vector<byte> devImgDst(ImgSize);

    //StartTimer(timerCUDA);
    func_diff_byte FuncDiff;
    thrust::transform(devImgSrc.begin(), devImgSrc.end(),
    		          devImgBack.begin(), devImgDst.begin(), FuncDiff);
    //StopTimer(timerCUDA);

    // transfer data back to host
    thrust::copy(devImgDst.begin(), devImgDst.end(), hostImgSrc.begin());

    // How to copy host_vectors to ImgDst efficient?
    for (idx = 0; idx < ImgSize; idx++)
    {
    	ImgDst[idx] = hostImgSrc[idx];
    }

    hipDeviceSynchronize();
    hipEventRecord(end,0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);

    return (elapsed_time); // Total time
    //return(GetTimer(timerCUDA));
};
