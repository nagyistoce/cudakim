#include "hip/hip_runtime.h"
/*
 * imageBackground.cu
 *
 * Finding background in a series of images in 3D, where the z-dimension is the time
 * the background image is found computing the median of the pixel intensity in the z-dimension
 *
 *  Created on: 26/09/2011
 *      Author: kimbjerge
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, bmp utilities
#include "defs.h"
#include "BmpUtil.h"
#include "timer.h"

static unsigned int timerCUDA = 0;

__global__ void
test3DImages (byte* dst, int stride, hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
	  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
	  byte* imgPtr = (byte *)devPitchedPtr.ptr;
	  size_t pitch = devPitchedPtr.pitch;
	  size_t slicePitch = pitch * height;
	  imgPtr += slicePitch*8;

	  byte cp = imgPtr[rowIdx * pitch + colIdx];

	  // Update average of images
	  dst[rowIdx * stride + colIdx] = cp;
}

// Sorts an array a of length depth
__device__ void insertionsort(byte *a, int depth)
{
	int i, j;
	byte t;
	for (i=1; i < depth; i++)
	{
		t = a[i];
		j = i-1;
		while(t < a[j] && j >= 0)
		{
			a[j+1] = a[j];
			j = j-1;
		}
		a[j+1] = t;
	}
}

__device__ void swap (byte *x, byte *y)
{
	byte tmp;
	tmp = *x;
	*x = *y;
	*y = tmp;
}

__device__ void bublesort (byte *a, int depth)
{
	int i, j;
	for (i = 0; i < (depth-1); i++)
		for (j = 0; j < (depth-(i+1)); j++)
			if (a[j] > a[j+1])
				swap(&a[j], &a[j+1]);
}


__global__ void
median3DImages (byte* dst, int stride, hipPitchedPtr devPitchedPtr, int width, int height, int depth)
{
	  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
	  byte* imgPtr = (byte *)devPitchedPtr.ptr;
	  size_t pitch = devPitchedPtr.pitch;
	  size_t slicePitch = pitch * height;
	  byte median[DEPTH];

	  // Average of all images
	  for (int z = 0; z < depth; ++z)
	  {
		  byte *slice = imgPtr + z * slicePitch; // Find sliced image
		  byte *row = slice + rowIdx * pitch; // Find row in image
		  median[z] = row[colIdx];
	  }

	  //insertionsort(median, depth); // NOT WORKING ON MAC MINI
	  bublesort(median, depth);

	  // Update average of images
	  dst[rowIdx * stride + colIdx] = median[(DEPTH+1)/2];
}

// Find background image based on 3D cube of images
float ImageBackground(byte *ImgDst, byte *ImgSrc, ROI Size, int Stride, int depth)
{
    byte *Dst;
    size_t DstStride;
    hipMemcpy3DParms memcpy3DParms = {0};

    DEBUG_MSG("[ImageBackground]\n");

    // Create src pointer and extent
    memcpy3DParms.srcPtr = make_hipPitchedPtr(ImgSrc, Stride, Size.width, Size.height);
    memcpy3DParms.extent = make_hipExtent(Size.width * sizeof(byte), Size.height, depth);

    // Allocation of memory for 3D source images in byte format
    cutilSafeCall(hipMalloc3D(&memcpy3DParms.dstPtr, memcpy3DParms.extent));

    DEBUG_MSG("srcPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.srcPtr.pitch, memcpy3DParms.srcPtr.xsize, memcpy3DParms.srcPtr.ysize);
    DEBUG_MSG("dstPtr: pitch, xsize, ysize [%d,%d,%d]\n", memcpy3DParms.dstPtr.pitch, memcpy3DParms.dstPtr.xsize, memcpy3DParms.dstPtr.ysize);

    // Copy images to device memory
    memcpy3DParms.kind = hipMemcpyHostToDevice;
    cutilSafeCall(hipMemcpy3D(&memcpy3DParms));

    // Allocation of memory for 2D destination image in byte format
    cutilSafeCall(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(byte), Size.height));

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid( ceil((float)Size.width / BLOCK_SIZE), ceil((float)Size.height / BLOCK_SIZE) );

    DEBUG_MSG("Grid (Blocks)    [%d,%d]\n", grid.x, grid.y);
    DEBUG_MSG("Threads in Block [%d,%d]\n", threads.x, threads.y);

    if (timerCUDA == 0) CreateTimer(&timerCUDA);
    RestartTimer(timerCUDA);

    median3DImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);
    //test3DImages<<< grid, threads >>>(Dst, DstStride, memcpy3DParms.dstPtr, Size.width, Size.height, depth);

    StopTimer(timerCUDA);

    cutilSafeCall(hipDeviceSynchronize());

    cutilSafeCall(hipMemcpy2D(ImgDst, Size.width * sizeof(byte),
                                Dst, DstStride * sizeof(byte),
                                Size.width * sizeof(byte), Size.height,
                                hipMemcpyDeviceToHost) );

    cutilSafeCall(hipFree(memcpy3DParms.dstPtr.ptr));

    return GetTimer(timerCUDA);;
}



